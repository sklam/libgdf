#include "hip/hip_runtime.h"
#include <gdf/gdf.h>
#include <gdf/utils.h>
#include <gdf/errorutils.h>


template<typename T, typename Tout, typename F>
__global__
void gpu_binary_op(const T *lhs_data, const gdf_valid_type *lhs_valid,
                   const T *rhs_data, const gdf_valid_type *rhs_valid,
                   gdf_size_type size, Tout *results, F functor) {
    int tid = threadIdx.x;
    int blkid = blockIdx.x;
    int blksz = blockDim.x;
    int gridsz = gridDim.x;

    int start = tid + blkid * blksz;
    int step = blksz * gridsz;
    if ( lhs_valid || rhs_valid ) {  // has valid mask
        for (int i=start; i<size; i+=step) {
            if (gdf_is_valid(lhs_valid, i) && gdf_is_valid(rhs_valid, i))
                results[i] = functor.apply(lhs_data[i], rhs_data[i]);
        }
    } else {                         // no valid mask
        for (int i=start; i<size; i+=step) {
            results[i] = functor.apply(lhs_data[i], rhs_data[i]);
        }
    }
}

template<typename T, typename Tout, typename F>
struct BinaryOp {
    static
    gdf_error launch(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
        if (lhs->size != rhs->size || lhs->size != output->size) {
            return GDF_COLUMN_SIZE_MISMATCH;
        }

        // find optimal blocksize
        int mingridsize, blocksize;
        CUDA_TRY(
            hipOccupancyMaxPotentialBlockSize(&mingridsize, &blocksize,
                                               gpu_binary_op<T, Tout, F>)
        );
        // find needed gridsize
        int gridsize = (lhs->size + blocksize - 1) / blocksize;

        F functor;
        gpu_binary_op<<<gridsize, blocksize>>>(
            // inputs
            (const T*)lhs->data, lhs->valid,
            (const T*)rhs->data, rhs->valid,
            lhs->size,
            // output
            (Tout*)output->data,
            // action
            functor
        );

        CUDA_CHECK_LAST();
        return GDF_SUCCESS;
    }
};

template<typename T, typename F>
struct ArithOp {
    static
    gdf_error launch(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
        return BinaryOp<T, T, F>::launch(lhs, rhs, output);
    }
};

template<typename T, typename F>
struct LogicalOp {
    static
    gdf_error launch(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
        return BinaryOp<T, int8_t, F>::launch(lhs, rhs, output);
    }
};


#define DEF_ARITH_OP_REAL(F)                                                  \
gdf_error F##_generic(gdf_column *lhs, gdf_column *rhs, gdf_column *output) { \
    switch ( lhs->dtype ) {                                                   \
    case GDF_FLOAT32: return F##_f32(lhs, rhs, output);                       \
    case GDF_FLOAT64: return F##_f64(lhs, rhs, output);                       \
    default: return GDF_UNSUPPORTED_DTYPE;                                    \
    }                                                                         \
}

#define DEF_ARITH_OP_NUM(F)                                                   \
gdf_error F##_generic(gdf_column *lhs, gdf_column *rhs, gdf_column *output) { \
    switch ( lhs->dtype ) {                                                   \
    case GDF_INT32:   return F##_i32(lhs, rhs, output);                       \
    case GDF_INT64:   return F##_i64(lhs, rhs, output);                       \
    case GDF_FLOAT32: return F##_f32(lhs, rhs, output);                       \
    case GDF_FLOAT64: return F##_f64(lhs, rhs, output);                       \
    default: return GDF_UNSUPPORTED_DTYPE;                                    \
    }                                                                         \
}

// Arithmeitc

template<typename T>
struct DeviceAdd {
    __device__
    T apply(T lhs, T rhs) {
        return lhs + rhs;
    }
};

template<typename T>
struct DeviceSub {
    __device__
    T apply(T lhs, T rhs) {
        return lhs - rhs;
    }
};

template<typename T>
struct DeviceMul {
    __device__
    T apply(T lhs, T rhs) {
        return lhs * rhs;
    }
};

template<typename T>
struct DeviceFloorDivInt {
    __device__
    T apply(T lhs, T rhs) {
        return std::floor((double)lhs / (double)rhs);
    }
};

template<typename T>
struct DeviceFloorDivReal {
    __device__
    T apply(T lhs, T rhs) {
        return std::floor(lhs / rhs);
    }
};

template<typename T>
struct DeviceDiv {
    __device__
    T apply(T lhs, T rhs) {
        return lhs / rhs;
    }
};

DEF_ARITH_OP_NUM(gdf_add)

gdf_error gdf_add_i32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return ArithOp<int32_t, DeviceAdd<int32_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_add_i64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return ArithOp<int64_t, DeviceAdd<int64_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_add_f32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return ArithOp<float, DeviceAdd<float> >::launch(lhs, rhs, output);
}

gdf_error gdf_add_f64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return ArithOp<double, DeviceAdd<double> >::launch(lhs, rhs, output);
}

DEF_ARITH_OP_NUM(gdf_sub)

gdf_error gdf_sub_i32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return ArithOp<int32_t, DeviceSub<int32_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_sub_i64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return ArithOp<int64_t, DeviceSub<int64_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_sub_f32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return ArithOp<float, DeviceSub<float> >::launch(lhs, rhs, output);
}

gdf_error gdf_sub_f64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return ArithOp<double, DeviceSub<double> >::launch(lhs, rhs, output);
}

DEF_ARITH_OP_NUM(gdf_mul)

gdf_error gdf_mul_i32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return ArithOp<int32_t, DeviceMul<int32_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_mul_i64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return ArithOp<int64_t, DeviceMul<int64_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_mul_f32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return ArithOp<float, DeviceMul<float> >::launch(lhs, rhs, output);
}

gdf_error gdf_mul_f64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return ArithOp<double, DeviceMul<double> >::launch(lhs, rhs, output);
}

DEF_ARITH_OP_NUM(gdf_floordiv)

gdf_error gdf_floordiv_i32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return ArithOp<int32_t, DeviceFloorDivInt<int32_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_floordiv_i64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return ArithOp<int64_t, DeviceFloorDivInt<int64_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_floordiv_f32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return ArithOp<float, DeviceFloorDivReal<float> >::launch(lhs, rhs, output);
}

gdf_error gdf_floordiv_f64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return ArithOp<double, DeviceFloorDivReal<double> >::launch(lhs, rhs, output);
}

DEF_ARITH_OP_REAL(gdf_div)

gdf_error gdf_div_f32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return ArithOp<float, DeviceDiv<float> >::launch(lhs, rhs, output);
}

gdf_error gdf_div_f64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return ArithOp<double, DeviceDiv<double> >::launch(lhs, rhs, output);
}


// logical


#define DEF_LOGICAL_OP_NUM(F)                                                 \
gdf_error F##_generic(gdf_column *lhs, gdf_column *rhs, gdf_column *output) { \
    if( output->dtype != GDF_INT8 ) return GDF_UNSUPPORTED_DTYPE;             \
    switch ( lhs->dtype ) {                                                   \
    case GDF_INT32:   return F##_i32(lhs, rhs, output);                       \
    case GDF_INT64:   return F##_i64(lhs, rhs, output);                       \
    case GDF_FLOAT32: return F##_f32(lhs, rhs, output);                       \
    case GDF_FLOAT64: return F##_f64(lhs, rhs, output);                       \
    default: return GDF_UNSUPPORTED_DTYPE;                                    \
    }                                                                         \
}

template<typename T>
struct DeviceGt {
    __device__
    bool apply(T lhs, T rhs) {
        return lhs > rhs;
    }
};

template<typename T>
struct DeviceGe {
    __device__
    bool apply(T lhs, T rhs) {
        return lhs >= rhs;
    }
};

template<typename T>
struct DeviceLt {
    __device__
    bool apply(T lhs, T rhs) {
        return lhs < rhs;
    }
};

template<typename T>
struct DeviceLe {
    __device__
    bool apply(T lhs, T rhs) {
        return lhs <= rhs;
    }
};

template<typename T>
struct DeviceEq {
    __device__
    bool apply(T lhs, T rhs) {
        return lhs == rhs;
    }
};


template<typename T>
struct DeviceNe {
    __device__
    bool apply(T lhs, T rhs) {
        return lhs != rhs;
    }
};

DEF_LOGICAL_OP_NUM(gdf_gt)

gdf_error gdf_gt_i32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<int32_t, DeviceGt<int32_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_gt_i64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<int64_t, DeviceGt<int64_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_gt_f32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<float, DeviceGt<float> >::launch(lhs, rhs, output);
}

gdf_error gdf_gt_f64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<double, DeviceGt<double> >::launch(lhs, rhs, output);
}

DEF_LOGICAL_OP_NUM(gdf_ge)

gdf_error gdf_ge_i32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<int32_t, DeviceGe<int32_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_ge_i64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<int64_t, DeviceGe<int64_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_ge_f32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<float, DeviceGe<float> >::launch(lhs, rhs, output);
}

gdf_error gdf_ge_f64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<double, DeviceGe<double> >::launch(lhs, rhs, output);
}


DEF_LOGICAL_OP_NUM(gdf_lt)

gdf_error gdf_lt_i32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<int32_t, DeviceLt<int32_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_lt_i64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<int64_t, DeviceLt<int64_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_lt_f32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<float, DeviceLt<float> >::launch(lhs, rhs, output);
}

gdf_error gdf_lt_f64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<double, DeviceLt<double> >::launch(lhs, rhs, output);
}

DEF_LOGICAL_OP_NUM(gdf_le)

gdf_error gdf_le_i32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<int32_t, DeviceLe<int32_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_le_i64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<int64_t, DeviceLe<int64_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_le_f32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<float, DeviceLe<float> >::launch(lhs, rhs, output);
}

gdf_error gdf_le_f64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<double, DeviceLe<double> >::launch(lhs, rhs, output);
}

DEF_LOGICAL_OP_NUM(gdf_eq)

gdf_error gdf_eq_i32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<int32_t, DeviceEq<int32_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_eq_i64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<int64_t, DeviceEq<int64_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_eq_f32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<float, DeviceEq<float> >::launch(lhs, rhs, output);
}

gdf_error gdf_eq_f64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<double, DeviceEq<double> >::launch(lhs, rhs, output);
}

DEF_LOGICAL_OP_NUM(gdf_ne)

gdf_error gdf_ne_i32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<int32_t, DeviceNe<int32_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_ne_i64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<int64_t, DeviceNe<int64_t> >::launch(lhs, rhs, output);
}

gdf_error gdf_ne_f32(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<float, DeviceNe<float> >::launch(lhs, rhs, output);
}

gdf_error gdf_ne_f64(gdf_column *lhs, gdf_column *rhs, gdf_column *output) {
    return LogicalOp<double, DeviceNe<double> >::launch(lhs, rhs, output);
}

